#include "hip/hip_runtime.h"
/******************************************************************************
 *cr
 *cr         (C) Copyright 2010-2013 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE


#include "defs.h"
#include "kernel_radix.cu"
#include "support.h"
#include<iostream>
using namespace std;
__constant__ unsigned short dc_flist_key_16_index[max_unique_items];
__global__ void histogram_kernel_naive(unsigned int* input, unsigned int* bins,
        unsigned int num_elements, unsigned int num_bins) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    while (i < num_elements) {
        int bin_num = input[i];
        if (bin_num < num_bins) {
            atomicAdd(&bins[bin_num], 1);
        }
        i+=stride;
    }
}
__global__ void histogram_kernel(unsigned int* input, unsigned int* bins,
        unsigned int num_elements) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int index_x = 0;
    extern __shared__ unsigned int hist_priv[];
    for (int i = 0; i < ceil(max_unique_items / (1.0 * blockDim.x)); i++){
        index_x = threadIdx.x + i * blockDim.x;
        if (index_x < max_unique_items )
            hist_priv[index_x] = 0;
    }

    __syncthreads();
    unsigned int stride = blockDim.x * gridDim.x;
    while (i < num_elements) {
        int bin_num = input[i];
        if (bin_num < max_unique_items ) {
            atomicAdd(&hist_priv[bin_num], 1);
        }
        i+=stride;
    }
    __syncthreads();
    for (int i = 0; i < ceil(max_unique_items / (1.0 * blockDim.x)); i++){
        index_x = threadIdx.x + i * blockDim.x;
        if (index_x < max_unique_items) {
            atomicAdd(&bins[index_x], hist_priv[index_x]);
        }
    }
}

    //make_flist(d_trans_offsets, d_transactions, d_flist, num_transactions, num_items_in_transactions);
void make_flist(unsigned int *d_trans_offset, unsigned int *d_transactions, unsigned int *d_flist,
        unsigned int num_transactions, unsigned int num_items_in_transactions, int SM_PER_BLOCK) {
    
    hipError_t cuda_ret;
    dim3 grid_dim, block_dim;
    block_dim.x = BLOCK_SIZE; 
    block_dim.y = 1; block_dim.z = 1;
    grid_dim.x = ceil(num_items_in_transactions / (16.0 * BLOCK_SIZE)); 
    grid_dim.y = 1; grid_dim.z = 1;
    if (max_unique_items * sizeof(unsigned int) < SM_PER_BLOCK) {
        // private histogram should fit in shared memory
        histogram_kernel<<<grid_dim, block_dim, max_unique_items * sizeof(unsigned int)>>>(d_transactions, d_flist, num_items_in_transactions);
    } else {
        // private histogram will not fit in shared memory. launch global kernel
        histogram_kernel_naive<<<grid_dim, block_dim>>>(d_transactions, d_flist, num_items_in_transactions, max_unique_items);
    }
    
    cuda_ret = hipDeviceSynchronize();
    if(cuda_ret != hipSuccess) FATAL("Unable to launch kernel");
}
    
   
   
   
#define INVALID 0XFFFFFF 
__global__ void sort_transaction_kernel(unsigned short *d_flist_key_16_index, unsigned int *d_flist, unsigned int *d_transactions,
        unsigned int *offset_array, unsigned int num_transactions, unsigned int num_elements, unsigned int bins, bool indexFileInConstantMem) {
   
    //unsigned int transaction_index = threadIdx.x + blockDim.x * blockIdx.x;
    //unsigned int stride = blockDim.x * gridDim.x;
    unsigned int transaction_start_index = blockDim.x * blockIdx.x;
    unsigned int transaction_end_index = transaction_start_index +  blockDim.x;
    //TBD: need to pass dynamically
    __shared__ unsigned int Ts[90][max_items_in_transaction];
    unsigned int index = threadIdx.x;
    
    __syncthreads();
    // clear SM 
    for (int i = 0; i < 90; i++) {
        while (index < max_items_in_transaction) {
            Ts[i][index] = INVALID;
            index += blockDim.x;
        }
        __syncthreads();
    }
    // get all the transaction assigned to this block into SM
    for (unsigned int i = transaction_start_index; i < transaction_end_index && i < num_transactions; i++) {
        // get the ith transaction data into SM
        int start_offset = offset_array[i];
        int end_offset = offset_array[i+1];
        int index1 = start_offset + threadIdx.x;
        __syncthreads();
        // threads collaborate to get the ith transaction
        while (index1 < end_offset) {
            Ts[i-transaction_start_index][index1 - start_offset] = d_transactions[index1];        
            index1 += blockDim.x;
        }
        __syncthreads();
    }

    // now that all transactions are in SM, each thread takes ownership of a row of SM
    // (i.e. one transaction per thread)
    if (threadIdx.x < 90) {
        for (int i =0; i < max_items_in_transaction;i++) {
            if (Ts[threadIdx.x][i] < INVALID) {
                Ts[threadIdx.x][i]++;
            } 
        }        
    }
    
    __syncthreads();
    // now that work is done write back results 
    for (unsigned int i = transaction_start_index; i < transaction_end_index && i < num_transactions; i++) {
        // get the ith transaction data from SM to global mem
        int start_offset = offset_array[i];
        int end_offset = offset_array[i+1];
        int index1 = start_offset + threadIdx.x;
        __syncthreads();
        while (index1 < end_offset) {
            d_transactions[index1] = Ts[i - transaction_start_index][index1 - start_offset];        
            index1 += blockDim.x;
        }
        __syncthreads();
    }
} 

void sort_transaction(unsigned short *d_flist_key_16_index, unsigned int *d_flist, unsigned int *d_transactions, unsigned int *offset_array, unsigned int num_transactions, unsigned int num_items_in_transactions, unsigned int bins,bool indexFileInConstantMem) {
    hipDeviceProp_t deviceProp;
    hipError_t ret;
    hipGetDeviceProperties(&deviceProp, 0);
    int SM_PER_BLOCK = deviceProp.sharedMemPerBlock;
    
    dim3 block_dim;
    dim3 grid_dim;
    
    unsigned int bytesPerTransaction = max_items_in_transaction * sizeof(unsigned int);
    
    block_dim.x = ((SM_PER_BLOCK / bytesPerTransaction) - 10) > 90 ? 90 : ((SM_PER_BLOCK / bytesPerTransaction) - 10);
    block_dim.y = 1;
    block_dim.y = 1;

    grid_dim.x = (int) ceil(num_transactions / (1.0 * block_dim.x));
    grid_dim.y = 1;
    grid_dim.z = 1;
#ifdef TEST_MODE
    cout<<"sort_transaction_kernel<bx,gx>"<<block_dim.x<<","<<grid_dim.x<<endl;
#endif
    sort_transaction_kernel<<<grid_dim, block_dim>>>(d_flist_key_16_index, d_flist, d_transactions, offset_array,
            num_transactions, num_items_in_transactions, bins, indexFileInConstantMem); 
    ret = hipDeviceSynchronize();
    if(ret != hipSuccess) FATAL("Unable to launch kernel");
    
    
}  
